#include "hip/hip_runtime.h"
#include <hip/hiprtc.h>
#include <vector>
#include <memory>
#include <unordered_map>
#include <iostream>
#include <type_traits>

#include "THC/THC.h"
#include "THC/THCApply.cuh"

struct Apply1Hash {
  std::string op;
  std::string type;
  int Adim;

  Apply1Hash(std::string op, std::string type, int Adim) : op(op), type(type), Adim(Adim) {}

  bool operator == (const Apply1Hash& other) const
  {
    return op == other.op && type == other.type && Adim == other.Adim;
  }
};

struct Apply2Hash {
  std::string op;
  std::string type;
  int Adim, Bdim;

  Apply2Hash(std::string op, std::string type, int Adim, int Bdim) : op(op), type(type), Adim(Adim), Bdim(Bdim) {}

  bool operator == (const Apply2Hash& other) const
  {
    return op == other.op && type == other.type && Adim == other.Adim && Bdim == other.Bdim;
  }
};

struct Apply3Hash {
  std::string op;
  std::string type;
  int Adim, Bdim, Cdim;

  Apply3Hash(std::string op, std::string type, int Adim, int Bdim, int Cdim) :
    op(op), type(type), Adim(Adim), Bdim(Bdim), Cdim(Cdim) {}

  bool operator == (const Apply3Hash& other) const
  {
    return op == other.op && type == other.type && Adim == other.Adim && Bdim == other.Bdim && Cdim == other.Cdim;
  }
};

namespace std
{
  template<>
  struct hash<Apply1Hash>
  {
    typedef Apply1Hash argument_type;
    typedef std::size_t result_type;

    result_type operator()(argument_type const& s) const
    {
      result_type const h1 ( std::hash<std::string>()(s.op));
      result_type const h2 ( std::hash<std::string>()(s.type));
      result_type const h3 ( std::hash<int>()(s.Adim));
      return (h1 ^ (h2 << 1)) ^ (h3 << 1);
    }
  };

  template<>
  struct hash<Apply2Hash>
  {
    typedef Apply2Hash argument_type;
    typedef std::size_t result_type;

    result_type operator()(argument_type const& s) const
    {
      result_type const h1 ( std::hash<std::string>()(s.op));
      result_type const h2 ( std::hash<std::string>()(s.type));
      result_type const h3 ( std::hash<int>()(s.Adim));
      result_type const h4 ( std::hash<int>()(s.Bdim));
      return ((h1 ^ (h2 << 1)) ^ (h3 << 1)) ^ (h4 << 1);
    }
  };

  template<>
  struct hash<Apply3Hash>
  {
    typedef Apply3Hash argument_type;
    typedef std::size_t result_type;

    result_type operator()(argument_type const& s) const
    {
      result_type const h1 ( std::hash<std::string>()(s.op));
      result_type const h2 ( std::hash<std::string>()(s.type));
      result_type const h3 ( std::hash<int>()(s.Adim));
      result_type const h4 ( std::hash<int>()(s.Bdim));
      result_type const h5 ( std::hash<int>()(s.Cdim));
      return (((h1 ^ (h2 << 1)) ^ (h3 << 1)) ^ (h4 << 1)) ^ (h5 << 1);
    }
  };
}

typedef std::vector<char> PTX;
typedef std::shared_ptr<PTX> PTXPtr;

typedef std::unordered_map<Apply1Hash, std::shared_ptr<PTX>> Apply1Cache;
typedef std::unordered_map<Apply2Hash, std::shared_ptr<PTX>> Apply2Cache;
typedef std::unordered_map<Apply3Hash, std::shared_ptr<PTX>> Apply3Cache;

Apply1Cache apply1cache;
Apply2Cache apply2cache;
Apply3Cache apply3cache;


inline void NVRTC_CHECK(hiprtcResult result)
{
  if(result != HIPRTC_SUCCESS)
    THError(hiprtcGetErrorString(result));
}

void compilePTX(const char* src,
    		const char* headers[],
		const char* includeNames[],
		std::vector<char>& ptx)
{
  hiprtcProgram program;
  NVRTC_CHECK(hiprtcCreateProgram(&program, src, NULL, 1, headers, includeNames));

  hiprtcResult result = hiprtcCompileProgram(program, 0, NULL); 
  if(result == HIPRTC_ERROR_COMPILATION)
  {
    size_t logsize;
    hiprtcGetProgramLogSize(program, &logsize);

    std::vector<char> log(logsize);
    hiprtcGetProgramLog(program, log.data());
    THError(log.data());
  }
  else
    NVRTC_CHECK(result);

  size_t ptx_size;
  NVRTC_CHECK(hiprtcGetCodeSize(program, &ptx_size));
  ptx.resize(ptx_size);
  NVRTC_CHECK(hiprtcGetCode(program, ptx.data()));
  NVRTC_CHECK(hiprtcDestroyProgram(&program));
}

inline void CUDA_CHECK(hipError_t result)
{
  if(result != hipSuccess)
  {
    const char* errstr;
    hipDrvGetErrorString(result, &errstr);
    THError(errstr);
  }
}

void launch(const char* ptx, const char* name, void* args[], dim3 grid, dim3 block, hipStream_t stream)
{
  hipModule_t module;
  hipFunction_t func;

  CUDA_CHECK(hipModuleLoadData(&module, ptx));
  CUDA_CHECK(hipModuleGetFunction(&func, module, name));

  CUDA_CHECK(hipModuleLaunchKernel(func,
                            grid.x, grid.y, grid.z,
                            block.x, block.y, block.z,
                            0, stream, args, NULL));

  CUDA_CHECK(hipModuleUnload(module));
}

// Example op: 'x = y*2'
const char* instanciate_apply1 = "                                      \n\
#define TYPE %s								\n\
#include <header.h>                                                     \n\
struct Op {                                                             \n\
  __device__ __forceinline__ void operator()(float* v) {                \n\
    float& x = *v;                                                      \n\
    %s;                                                                 \n\
  }                                                                     \n\
};                                                                      \n\
extern \"C\" __global__                                                 \n\
void kernel(TensorInfo<TYPE> a, TYPE totalElements)                     \n\
{                                                                       \n\
  Op op;                                                                \n\
  THCudaTensor_pointwiseApply1<Op,TYPE,%d> (a, totalElements, op);	\n\
}                                                                       \n\
";

template <typename IndexType>
void THCudaTensor_pointwiseApply1RTC(
    TensorInfo<IndexType> aInfo,
    const char* apply_header,
    const char* op,
    IndexType totalElements,
    dim3 grid, dim3 block,
    int A,
    hipStream_t stream)
{
  // using c++11 std::is_same here
  const char* type;
  if (std::is_same<IndexType, unsigned long>::value)
    type = "unsigned long";
  else if(std::is_same<IndexType, unsigned int>::value)
    type = "unsigned int";

  char src[2048];
  sprintf(src, instanciate_apply1, type, op, A);
  const char *headers[] = {apply_header};
  const char *includeNames[] = {"header.h"};

  PTXPtr ptx;
  Apply1Hash hash(op, type, A);
  auto found_hash = apply1cache.find(hash);
  if(found_hash == apply1cache.end())
  {
    ptx = PTXPtr(new PTX());
    compilePTX(src, headers, includeNames, *ptx);
    apply1cache.emplace(hash, ptx);
  }
  else
    ptx = found_hash->second;

  void *args[] = {(void*)&aInfo, (void*)&totalElements};
  launch(ptx->data(), "kernel", args, grid, block, (hipStream_t)stream);
}

// Example op: 'x = x*y'
const char* instanciate_apply2 = "                                      \n\
#define TYPE %s								\n\
#include <header.h>                                                     \n\
struct Op {                                                             \n\
  __device__ __forceinline__						\n\
  void operator()(float* a, float* b) {      				\n\
    float& x = *a;                                                      \n\
    float& y = *b;							\n\
    %s;                                                                 \n\
  }                                                                     \n\
};                                                                      \n\
extern \"C\" __global__                                                 \n\
void kernel(TensorInfo<TYPE> a, TensorInfo<TYPE> b, TYPE totalElements) \n\
{                                                                       \n\
  Op op;                                                                \n\
  THCudaTensor_pointwiseApply2<Op,TYPE,%d,%d>				\n\
  			(a, b, totalElements, op);			\n\
}                                                                       \n\
";

template <typename IndexType>
void THCudaTensor_pointwiseApply2RTC(
    TensorInfo<IndexType> aInfo,
    TensorInfo<IndexType> bInfo,
    const char* apply_header,
    const char* op,
    IndexType totalElements,
    dim3 grid, dim3 block,
    int A, int B,
    hipStream_t stream)
{
  // using c++11 std::is_same here
  const char* type;
  if (std::is_same<IndexType, unsigned long>::value)
    type = "unsigned long";
  else if(std::is_same<IndexType, unsigned int>::value)
    type = "unsigned int";

  char src[2048];
  sprintf(src, instanciate_apply2, type, op, A, B);
  const char *headers[] = {apply_header};
  const char *includeNames[] = {"header.h"};

  PTXPtr ptx;
  Apply2Hash hash(op, type, A, B);
  auto found_hash = apply2cache.find(hash);
  if(found_hash == apply2cache.end())
  {
    ptx = PTXPtr(new PTX());
    compilePTX(src, headers, includeNames, *ptx);
    apply2cache.emplace(hash, ptx);
  }
  else
    ptx = found_hash->second;

  void *args[] = {(void*)&aInfo, (void*)&bInfo, (void*)&totalElements};
  launch(ptx->data(), "kernel", args, grid, block, (hipStream_t)stream);
}


// Example op: 'x = y*z'
const char* instanciate_apply3 = "                                      \n\
#define TYPE %s								\n\
#include <header.h>                                                     \n\
struct Op {                                                             \n\
  __device__ __forceinline__						\n\
  void operator()(float* a, float* b, float *c) {			\n\
    float& x = *a;                                                      \n\
    float& y = *b;							\n\
    float& z = *c;							\n\
    %s;                                                                 \n\
  }                                                                     \n\
};                                                                      \n\
extern \"C\" __global__                                                 \n\
void kernel(TensorInfo<TYPE> a,						\n\
    	    TensorInfo<TYPE> b,						\n\
    	    TensorInfo<TYPE> c,						\n\
	    TYPE totalElements)       					\n\
{                                                                       \n\
  Op op;                                                                \n\
  THCudaTensor_pointwiseApply3<Op,TYPE,%d,%d,%d>			\n\
  			(a, b, c, totalElements, op);			\n\
}                                                                       \n\
";

template <typename IndexType>
void THCudaTensor_pointwiseApply3RTC(
    TensorInfo<IndexType> aInfo,
    TensorInfo<IndexType> bInfo,
    TensorInfo<IndexType> cInfo,
    const char* apply_header,
    const char* op,
    IndexType totalElements,
    dim3 grid, dim3 block,
    int A, int B, int C,
    hipStream_t stream)
{
  // using c++11 std::is_same here
  const char* type;
  if (std::is_same<IndexType, unsigned long>::value)
    type = "unsigned long";
  else if(std::is_same<IndexType, unsigned int>::value)
    type = "unsigned int";

  char src[2048];
  sprintf(src, instanciate_apply3, type, op, A, B, C);
  const char *headers[] = {apply_header};
  const char *includeNames[] = {"header.h"};

  PTXPtr ptx;
  Apply3Hash hash(op, type, A, B, C);
  auto found_hash = apply3cache.find(hash);
  if(found_hash == apply3cache.end())
  {
    ptx = PTXPtr(new PTX());
    compilePTX(src, headers, includeNames, *ptx);
    apply3cache.emplace(hash, ptx);
  }
  else
    ptx = found_hash->second;

  void *args[] = {(void*)&aInfo, (void*)&bInfo, (void*)&cInfo, (void*)&totalElements};
  launch(ptx->data(), "kernel", args, grid, block, (hipStream_t)stream);
}


extern "C" 
bool THCudaTensor_pointwiseApply1(THCState* state,
                                  THCudaTensor* a,
                                  const char* apply_header,
                                  const char* op_string)
{
  TensorArgType aType = ReadWrite;
  hipStream_t stream = state->currentStream;
  long totalElements = THCudaTensor_nElement(state, a);

  if (THCudaTensor_nDimension(state, a) > MAX_CUTORCH_DIMS) {
    return false;
  }

  if (THCudaTensor_nDimension(state, a) == 0) {
    // Zero-dim tensor; do nothing
    return true;
  }

  const dim3 block = getApplyBlock();

  dim3 grid;
  if (!getApplyGrid(state, totalElements, grid)) {
    return false;
  }

  // If tensor args have overlapping indices and are read/write, then
  // we must expand the tensor to a contiguous form first, since
  // otherwise there are conflicting writes. Upon copying back to the
  // non-contiguous form, there will be conflicting writes, but at
  // least with copy, one of the updaters will win atomically. This is
  // a sketchy property of the old system as well (writing into all
  // indices of a tensor with overlapping indices should probably be
  // an error, since it is unclear which one should win), but we will
  // preserve this last-writer-wins (in arbitrary copy order) behavior.
  THCudaTensor* oldA = NULL;

  if (aType == ReadWrite && THC_overlappingIndices(state, a)) {
    // Must perform in contiguous space
    oldA = a;
    a = THCudaTensor_newContiguous(state, a);
  }

  // It is possible that the tensor dimensions are able to be collapsed,
  // and thus we can reduce the actual code complexity of the copy by
  // exploiting this knowledge statically, since the div/mod is the
  // most expensive part of the operation, more so than memory accesses.
  // For instance, when copying a non-contiguous to a contiguous tensor
  // (or vice versa), the contiguous tensor can be collapsed to one
  // dimension, and the loop to translate the linear index to the array
  // index can be similarly collapsed. That is what this unrolling is for.
#define HANDLE_CASE(TYPE, A)                                   \
  THCudaTensor_pointwiseApply1RTC(aInfo, apply_header, op_string, (TYPE)totalElements, grid, block, A, stream);

#define HANDLE_A_CASE(TYPE, A)                      \
  {                                                 \
    if (aInfo.isContiguous()) {                     \
      HANDLE_CASE(TYPE, -2);                        \
    } else {                                        \
      switch (A) {                                  \
        case 1:                                     \
          HANDLE_CASE(TYPE, 1);                     \
          break;                                    \
        case 2:                                     \
          HANDLE_CASE(TYPE, 2);                     \
          break;                                    \
        case 3:                                     \
          HANDLE_CASE(TYPE, 3);                     \
          break;                                    \
        default:                                    \
          HANDLE_CASE(TYPE, -1);                    \
          break;                                    \
      }                                             \
    }                                               \
  }

  // Can we use 32-bit integer math in the kernel (the linear ID for the copy
  // and the resulting non-linear offset is all computable using 32-bit math?)
  // We also use unsigned index math in the kernel, as signed div/mod has
  // additional overhead.
  if (THC_canUse32BitIndexMath(state, a)) {
    TensorInfo<unsigned int> aInfo(state, a);

    HANDLE_A_CASE(unsigned int, aInfo.dims);
  } else {
    TensorInfo<unsigned long> aInfo(state, a);

    // For large tensors, we only compile the completely contiguous
    // version and the completely generic version, to reduce
    // compilation time.
    if (aInfo.isContiguous()) {
      THCudaTensor_pointwiseApply1RTC(aInfo, apply_header, op_string, (unsigned long)totalElements, grid, block, -2, stream);
    } else {
      THCudaTensor_pointwiseApply1RTC(aInfo, apply_header, op_string, (unsigned long)totalElements, grid, block, -1, stream);
    }
  }
#undef HANDLE_CASE
#undef HANDLE_A_CASE

  if (oldA) {
    // Ignore overlaps when copying back; if we use THCudaTensor_copy
    // instead, it will recursively try and invoke ourselves to make
    // oldA contiguous.
    THCudaTensor_copyIgnoringOverlaps(state, oldA, a);
    THCudaTensor_free(state, a);
    a = oldA;
  }

  return true;
}

extern "C"
bool THCudaTensor_pointwiseApply2(THCState* state,
                                  THCudaTensor* a,
                                  THCudaTensor* b,
                                  const char* apply_header,
                                  const char* op_string)
{
  TensorArgType aType = ReadWrite;
  TensorArgType bType = ReadWrite;
  hipStream_t stream = state->currentStream;

  long totalElements = THCudaTensor_nElement(state, a);

  if (totalElements != THCudaTensor_nElement(state, b)) {
    return false;
  }

  if (THCudaTensor_nDimension(state, a) > MAX_CUTORCH_DIMS ||
      THCudaTensor_nDimension(state, b) > MAX_CUTORCH_DIMS) {
    return false;
  }

  if (THCudaTensor_nDimension(state, a) == 0) {
    // Zero-dim tensor; do nothing
    return true;
  }

  const dim3 block = getApplyBlock();

  dim3 grid;
  if (!getApplyGrid(state, totalElements, grid)) {
    return false;
  }

  // If tensor args have overlapping indices and are read/write, then
  // we must expand the tensor to a contiguous form first, since
  // otherwise there are conflicting writes. Upon copying back to the
  // non-contiguous form, there will be conflicting writes, but at
  // least with copy, one of the updaters will win atomically. This is
  // a sketchy property of the old system as well (writing into all
  // indices of a tensor with overlapping indices should probably be
  // an error, since it is unclear which one should win), but we will
  // preserve this last-writer-wins (in arbitrary copy order) behavior.
  THCudaTensor* oldA = NULL;
  THCudaTensor* oldB = NULL;

  if (aType == ReadWrite && THC_overlappingIndices(state, a)) {
    // Must perform in contiguous space
    oldA = a;
    a = THCudaTensor_newContiguous(state, a);
  }
  if (bType == ReadWrite && THC_overlappingIndices(state, b)) {
    // Must perform in contiguous space
    oldB = b;
    b = THCudaTensor_newContiguous(state, b);
  }

  // It is possible that the tensor dimensions are able to be collapsed,
  // and thus we can reduce the actual code complexity of the copy by
  // exploiting this knowledge statically, since the div/mod is the
  // most expensive part of the operation, more so than memory accesses.
  // For instance, when copying a non-contiguous to a contiguous tensor
  // (or vice versa), the contiguous tensor can be collapsed to one
  // dimension, and the loop to translate the linear index to the array
  // index can be similarly collapsed. That is what this unrolling is for.
#define HANDLE_CASE(TYPE, A, B)                                \
  THCudaTensor_pointwiseApply2RTC(aInfo, bInfo, apply_header, op_string, (TYPE)totalElements, grid, block, A, B, stream);

#define HANDLE_B_CASE(TYPE, A, B)                   \
  {                                                 \
    if (bInfo.isContiguous()) {                     \
      HANDLE_CASE(TYPE, A, -2);                     \
    } else {                                        \
      switch (B) {                                  \
        case 1:                                     \
          HANDLE_CASE(TYPE, A, 1);                  \
          break;                                    \
        case 2:                                     \
          HANDLE_CASE(TYPE, A, 2);                  \
          break;                                    \
        case 3:                                     \
          HANDLE_CASE(TYPE, A, 3);                  \
          break;                                    \
        default:                                    \
          HANDLE_CASE(TYPE, A, -1);                 \
          break;                                    \
      }                                             \
    }                                               \
  }

#define HANDLE_A_CASE(TYPE, A, B)                   \
  {                                                 \
    if (aInfo.isContiguous()) {                     \
      HANDLE_B_CASE(TYPE, -2, B);                   \
    } else {                                        \
      switch (A) {                                  \
        case 1:                                     \
          HANDLE_B_CASE(TYPE, 1, B);                \
          break;                                    \
        case 2:                                     \
          HANDLE_B_CASE(TYPE, 2, B);                \
          break;                                    \
        case 3:                                     \
          HANDLE_B_CASE(TYPE, 3, B);                \
          break;                                    \
        default:                                    \
          HANDLE_B_CASE(TYPE, -1, B);               \
          break;                                    \
      }                                             \
    }                                               \
  }

  if (THC_canUse32BitIndexMath(state, a) &&
      THC_canUse32BitIndexMath(state, b)) {
    TensorInfo<unsigned int> aInfo(state, a);
    TensorInfo<unsigned int> bInfo(state, b);

    HANDLE_A_CASE(unsigned int, aInfo.dims, bInfo.dims);
  } else {
    TensorInfo<unsigned long> aInfo(state, a);
    TensorInfo<unsigned long> bInfo(state, b);

    // For large tensors, we only compile the completely contiguous
    // version and the completely generic version, to reduce
    // compilation time.
    if (aInfo.isContiguous() && bInfo.isContiguous()) {
      THCudaTensor_pointwiseApply2RTC(aInfo, bInfo, apply_header, op_string,
	  			(unsigned long)totalElements, grid, block, -2, -2, stream);
    } else {
      THCudaTensor_pointwiseApply2RTC(aInfo, bInfo, apply_header, op_string,
	  			(unsigned long)totalElements, grid, block, -1, -1, stream);
    }
  }
#undef HANDLE_CASE
#undef HANDLE_B_CASE
#undef HANDLE_A_CASE

  if (oldA) {
    // Ignore overlaps when copying back; if we use THCudaTensor_copy
    // instead, it will recursively try and invoke ourselves to make
    // oldA contiguous.
    THCudaTensor_copyIgnoringOverlaps(state, oldA, a);
    THCudaTensor_free(state, a);
    a = oldA;
  }

  if (oldB) {
    // Ignore overlaps when copying back; if we use THCudaTensor_copy
    // instead, it will recursively try and invoke ourselves to make
    // oldB contiguous.
    THCudaTensor_copyIgnoringOverlaps(state, oldB, b);
    THCudaTensor_free(state, b);
    b = oldB;
  }

  return true;
}
extern "C"
bool THCudaTensor_pointwiseApply3(THCState* state,
                                  THCudaTensor* a,
                                  THCudaTensor* b,
                                  THCudaTensor* c,
                                  const char* apply_header,
                                  const char* op_string)
{
  TensorArgType aType = ReadWrite;
  TensorArgType bType = ReadWrite;
  TensorArgType cType = ReadWrite;
  hipStream_t stream = state->currentStream;

  long totalElements = THCudaTensor_nElement(state, a);

  if (totalElements != THCudaTensor_nElement(state, b) ||
      totalElements != THCudaTensor_nElement(state, c)) {
    return false;
  }

  if (THCudaTensor_nDimension(state, a) > MAX_CUTORCH_DIMS ||
      THCudaTensor_nDimension(state, b) > MAX_CUTORCH_DIMS ||
      THCudaTensor_nDimension(state, c) > MAX_CUTORCH_DIMS) {
    return false;
  }

  if (THCudaTensor_nDimension(state, a) == 0) {
    // Zero-dim tensor; do nothing
    return true;
  }

  const dim3 block = getApplyBlock();

  dim3 grid;
  if (!getApplyGrid(state, totalElements, grid)) {
    return false;
  }

  // If tensor args have overlapping indices and are read/write, then
  // we must expand the tensor to a contiguous form first, since
  // otherwise there are conflicting writes. Upon copying back to the
  // non-contiguous form, there will be conflicting writes, but at
  // least with copy, one of the updaters will win atomically. This is
  // a sketchy property of the old system as well (writing into all
  // indices of a tensor with overlapping indices should probably be
  // an error, since it is unclear which one should win), but we will
  // preserve this last-writer-wins (in arbitrary copy order) behavior.
  THCudaTensor* oldA = NULL;
  THCudaTensor* oldB = NULL;
  THCudaTensor* oldC = NULL;

  if (aType == ReadWrite && THC_overlappingIndices(state, a)) {
    // Must perform in contiguous space
    oldA = a;
    a = THCudaTensor_newContiguous(state, a);
  }

  if (bType == ReadWrite && THC_overlappingIndices(state, b)) {
    // Must perform in contiguous space
    oldB = b;
    b = THCudaTensor_newContiguous(state, b);
  }

  if (cType == ReadWrite && THC_overlappingIndices(state, c)) {
    // Must perform in contiguous space
    oldC = c;
    c = THCudaTensor_newContiguous(state, c);
  }

#define HANDLE_CASE(TYPE, A, B, C)                                      \
  THCudaTensor_pointwiseApply3RTC(aInfo, bInfo, cInfo,			\
      apply_header, op_string, (TYPE)totalElements, grid, block,	\
      A, B, C, stream);								\

#define HANDLE_C_CASE(TYPE, A, B, C)             \
  {                                              \
    if (cInfo.isContiguous()) {                  \
      HANDLE_CASE(TYPE, A, B, -2);               \
    } else {                                     \
      switch (C) {                               \
        case 1:                                  \
          HANDLE_CASE(TYPE, A, B, 1);            \
          break;                                 \
        case 2:                                  \
          HANDLE_CASE(TYPE, A, B, 2);            \
          break;                                 \
        case 3:                                  \
          HANDLE_CASE(TYPE, A, B, 3);            \
          break;                                 \
        default:                                 \
          HANDLE_CASE(TYPE, A, B, -1);           \
          break;                                 \
      }                                          \
    }                                            \
  }

#define HANDLE_B_CASE(TYPE, A, B, C)                 \
  {                                                  \
    if (bInfo.isContiguous()) {                      \
      HANDLE_C_CASE(TYPE, A, -2, C);                 \
    } else {                                         \
      switch (B) {                                   \
        case 1:                                      \
          HANDLE_C_CASE(TYPE, A, 1, C);              \
          break;                                     \
        case 2:                                      \
          HANDLE_C_CASE(TYPE, A, 2, C);              \
          break;                                     \
        case 3:                                      \
          HANDLE_C_CASE(TYPE, A, 3, C);              \
          break;                                     \
        default:                                     \
          HANDLE_C_CASE(TYPE, A, -1, C);             \
          break;                                     \
      }                                              \
    }                                                \
  }

#define HANDLE_A_CASE(TYPE, A, B, C)                 \
  {                                                  \
    if (aInfo.isContiguous()) {                      \
      HANDLE_B_CASE(TYPE, -2, B, C);                 \
    } else {                                         \
      switch (A) {                                   \
        case 1:                                      \
          HANDLE_B_CASE(TYPE, 1, B, C);              \
          break;                                     \
        case 2:                                      \
          HANDLE_B_CASE(TYPE, 2, B, C);              \
          break;                                     \
        case 3:                                      \
          HANDLE_B_CASE(TYPE, 3, B, C);              \
          break;                                     \
        default:                                     \
          HANDLE_B_CASE(TYPE, -1, B, C);             \
          break;                                     \
      }                                              \
    }                                                \
  }

  if (THC_canUse32BitIndexMath(state, a) &&
      THC_canUse32BitIndexMath(state, b) &&
      THC_canUse32BitIndexMath(state, c)) {
    TensorInfo<unsigned int> aInfo(state, a);
    TensorInfo<unsigned int> bInfo(state, b);
    TensorInfo<unsigned int> cInfo(state, c);

    HANDLE_A_CASE(unsigned int, aInfo.dims, bInfo.dims, cInfo.dims);
  } else {
    TensorInfo<unsigned long> aInfo(state, a);
    TensorInfo<unsigned long> bInfo(state, b);
    TensorInfo<unsigned long> cInfo(state, c);

    // For large tensors, we only compile the completely contiguous
    // version and the completely generic version, to reduce
    // compilation time.
    if (aInfo.isContiguous() && bInfo.isContiguous() && cInfo.isContiguous()) {
      THCudaTensor_pointwiseApply3RTC(aInfo, bInfo, cInfo,
	  		apply_header, op_string,
	  		(unsigned long)totalElements, grid, block, -2, -2, -2, stream);
    } else {
      THCudaTensor_pointwiseApply3RTC(aInfo, bInfo, cInfo,
	  		apply_header, op_string,
	  		(unsigned long)totalElements, grid, block, -1, -1, -1, stream);
    }
  }
#undef HANDLE_CASE
#undef HANDLE_C_CASE
#undef HANDLE_B_CASE
#undef HANDLE_A_CASE

  if (oldA) {
    // Ignore overlaps when copying back; if we use THCudaTensor_copy
    // instead, it will recursively try and invoke ourselves to make
    // oldA contiguous.
    THCudaTensor_copyIgnoringOverlaps(state, oldA, a);
    THCudaTensor_free(state, a);
    a = oldA;
  }

  if (oldB) {
    // Ignore overlaps when copying back; if we use THCudaTensor_copy
    // instead, it will recursively try and invoke ourselves to make
    // oldB contiguous.
    THCudaTensor_copyIgnoringOverlaps(state, oldB, b);
    THCudaTensor_free(state, b);
    b = oldB;
  }

  if (oldC) {
    // Ignore overlaps when copying back; if we use THCudaTensor_copy
    // instead, it will recursively try and invoke ourselves to make
    // oldC contiguous.
    THCudaTensor_copyIgnoringOverlaps(state, oldC, c);
    THCudaTensor_free(state, c);
    c = oldC;
  }

  return true;
}
